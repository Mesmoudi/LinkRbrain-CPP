#include <stdio.h>
#include <hip/hip_runtime.h>


#define CHECK_CUDA_ERROR(cuda_result) \
    if ((int)cuda_result != 0) { \
        fprintf(stderr, "CUDA native error at line %d in %s : %s - %s\n", __LINE__, __FILE__, hipGetErrorName(cuda_result), hipGetErrorString(cuda_result)); \
    }
//
// #define CHECK_CUDA_ERROR(cuda_result) \
//     if ((int)cuda_result != 0) { \
//         const char* error_name; \
//         const char* error_string; \
//         cuGetErrorName(cuda_result, &error_name); \
//         cuGetErrorString(cuda_result, &error_string); \
//         fprintf(stderr, "CUDA native error: %s - %s", error_name, error_string); \
//     }


__global__ void CUDA_test(float a, float* in1, float* in2, float* out, size_t n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        const float d = in1[i] - in2[i];
        const float dd = (i == 0) ? 0.0 : (in1[i+1] - in1[i]);
        out[i] = a * sqrt(in1[i] + in2[i] / a) / in1[i] + in1[i] / in2[i] - a;
        out[i] += d*d + dd*dd;
        out[i] -= a / d + dd - sqrt(a);
        for (float s=0.f; s<10.f; ++s) {
            out[i] += ++s;
        }
        // printf("thread %d, block %d, index %d, value %f\n", threadIdx.x, blockIdx.x, i, out[i]);
    }
}


class Calculator {
public:

    Calculator() {
        _n = 0;
        _threads = 128;
        //  CHECK_CUDA_ERROR(cudaDeviceSetCacheConfig(cudaFuncCachePreferL1));
        //  CHECK_CUDA_ERROR(cudaDeviceSetCacheConfig(cudaFuncCachePreferShared));
        //  CHECK_CUDA_ERROR(cudaSetDeviceFlags(cudaDeviceScheduleBlockingSync | cudaDeviceMapHost));
        CHECK_CUDA_ERROR(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
    }
    ~Calculator() {
        deallocate();
    }

    void allocate(size_t n) {
        if (n != _n) {
            deallocate();
            _n = n;
            _size = _n * sizeof(float);
            CHECK_CUDA_ERROR(hipMalloc((void**) &_in1, _size));
            CHECK_CUDA_ERROR(hipMalloc((void**) &_in2, _size));
            CHECK_CUDA_ERROR(hipMalloc((void**) &_out, _size));
        }
    }
    void deallocate() {
        if (_n != 0) {
            CHECK_CUDA_ERROR(hipFree((void*) _in1));
            CHECK_CUDA_ERROR(hipFree((void*) _in2));
            CHECK_CUDA_ERROR(hipFree((void*) _out));
        }
    }

    void test(float a, float* in1, float* in2, float* out, size_t n) {
        allocate(n);
        CHECK_CUDA_ERROR(hipMemcpy(_in1, in1, _size, hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(_in2, in2, _size, hipMemcpyHostToDevice));
        size_t blocks = ceil(n / _threads);
        CUDA_test<<<blocks, _threads>>>(a, _in1, _in2, _out, n);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        CHECK_CUDA_ERROR(hipMemcpy(out, _out, _size, hipMemcpyDeviceToHost));
    }

    void set_threads(size_t threads) {
        _threads = threads;
    }

private:
    size_t _threads;
    size_t _n;
    size_t _size;
    float* _in1;
    float* _in2;
    float* _out;
};



static Calculator calculator;

void set_gpu_threads(size_t threads) {
    calculator.set_threads(threads);
}
void test_gpu(float a, float* in1, float* in2, float* out, size_t n) {
    calculator.test(a, in1, in2, out, n);
}
