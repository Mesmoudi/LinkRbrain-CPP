#include <vector>
#include <iostream>

namespace Types {
    #pragma pack(push,1)
    template <typename T>
    struct Point {
        T x, y, z, weight;
    };
    #pragma pack(pop)
}


#include <hip/hip_runtime.h>

template <typename T>
__global__ void _score_precompute_sphere(T* _precomputed, const Types::Point<T> p1, const size_t _groups_count, const Types::Point<T>* _groups_points, const size_t* _groups_offsets, const double diameter) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= _groups_count) return;
    size_t offset_min = _groups_offsets[i];
    size_t offset_max = _groups_offsets[i + 1];
    T score = 0.0;
    for (size_t offset=offset_min; offset<offset_max; ++offset) {
        Types::Point<T> p2 = _groups_points[offset];
        // const T dx = p1.x - p2.x;
        // const T dy = p1.y - p2.y;
        // const T dz = p1.z - p2.z;
        // const T distance = sqrt(dx*dx + dy*dy + dz*dz);
        // const T distance = norm3d(p1.x - p2.x, p1.y - p2.y, p1.z - p2.z);
        // const T x = distance / diameter;
        const T x = norm3d(p1.x - p2.x, p1.y - p2.y, p1.z - p2.z) / diameter;
        if (x < 1.0) {
            score += sqrt(p1.weight * p2.weight) * (0.5*x * (x*x - 3.0) + 1.0);
        }
    }
    _precomputed[i] = score;
}


#define CHECK_CUDA_ERROR(cuda_result) \
    if ((int)cuda_result != 0) { \
        std::cerr << "CUDA native error at line " << __LINE__ << " in " << __FILE__ << " : " << hipGetErrorName(cuda_result) << ", " << hipGetErrorString(cuda_result) << "\n"; \
        exit(1); \
    }


template <typename T>
class Precomputer {
public:

    Precomputer(const int device, const size_t threads) :
        _device(device),
        _threads(threads),
        _groups_count(0),
        _groups_points(NULL),
        _groups_offsets(NULL),
        _precomputed(NULL)
    {
        int count;
        CHECK_CUDA_ERROR(hipGetDeviceCount(&count));
        CHECK_CUDA_ERROR(hipSetDevice(_device));
        CHECK_CUDA_ERROR(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
        CHECK_CUDA_ERROR(hipGetDeviceProperties(&_gpu_properties, _device));
        if (threads > _gpu_properties.maxThreadsPerBlock) {
            _threads = _gpu_properties.maxThreadsPerBlock;
        }
        std::cout << count << " device(s)\n";
        std::cout << _gpu_properties.name << " selected\n";
        std::cout << _gpu_properties.totalGlobalMem << " bytes available (global memory)\n";
        std::cout << _gpu_properties.totalConstMem << " bytes available (constant memory)\n";
        std::cout << _gpu_properties.maxThreadsPerBlock << " threads maximum\n";
        std::cout << _gpu_properties.multiProcessorCount << " multiprocessors\n";
    }
    ~Precomputer() {
        deallocate();
    }

    void deallocate() {
        if (_groups_points != NULL) {
            CHECK_CUDA_ERROR(hipFree((void*) _groups_points));
            _groups_points = NULL;
        }
        if (_groups_offsets != NULL) {
            CHECK_CUDA_ERROR(hipFree((void*) _groups_offsets));
            _groups_offsets = NULL;
        }
        if (_precomputed != NULL) {
            CHECK_CUDA_ERROR(hipFree((void*) _precomputed));
            _precomputed = NULL;
        }
    }

    void set_threads(const size_t threads) {
        _threads = threads;
        _blocks = ceil(_groups_count / _threads);
    }

    void set_groups(const size_t groups_count, const std::vector<Types::Point<T>>& groups_points, const std::vector<size_t>& groups_offsets) {
        deallocate();
        _groups_count = groups_count;
        _blocks = ceil(groups_count / _threads);
        CHECK_CUDA_ERROR(hipMalloc((void**) &_groups_points, groups_points.size() * sizeof(Types::Point<T>)));
        CHECK_CUDA_ERROR(hipMemcpy(_groups_points, groups_points.data(), groups_points.size() * sizeof(Types::Point<T>), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMalloc((void**) &_groups_offsets, groups_offsets.size() * sizeof(size_t)));
        CHECK_CUDA_ERROR(hipMemcpy(_groups_offsets, groups_offsets.data(), groups_offsets.size() * sizeof(size_t), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMalloc((void**) &_precomputed, groups_count * sizeof(T)));
    }

    void precompute_sphere(std::vector<T>& precomputed, const Types::Point<T> center, const double diameter) {
        _score_precompute_sphere<T><<<_blocks, _threads>>>(
            _precomputed,
            center,
            _groups_count,
            _groups_points,
            _groups_offsets,
            diameter
        );
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        CHECK_CUDA_ERROR(hipMemcpy(&(precomputed[0]), _precomputed, _groups_count * sizeof(T), hipMemcpyDeviceToHost));
    }

private:

    const int _device;
    hipDeviceProp_t _gpu_properties;
    size_t _blocks;
    size_t _threads;
    //
    T* _precomputed;
    size_t _groups_count;
    Types::Point<T>* _groups_points;
    size_t* _groups_offsets;
};

template <typename T>
static Precomputer<T>* precomputer;

void CUDA_precomputing_start(const int device, const size_t threads) {
    precomputer<double> = new Precomputer<double>(device, threads);
}
void CUDA_precomputing_set_groups(const size_t groups_count, const std::vector<Types::Point<double>>& groups_points, const std::vector<size_t>& groups_offsets) {
    precomputer<double>->set_groups(groups_count, groups_points, groups_offsets);
}
void CUDA_precomputing_precompute_sphere(std::vector<double>& precomputed, const Types::Point<double> center, const double diameter) {
    precomputer<double>->precompute_sphere(precomputed, center, diameter);
}
void CUDA_precomputing_finish() {
    delete precomputer<double>;
}
